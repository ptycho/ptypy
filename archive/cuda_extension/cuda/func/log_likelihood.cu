#include "hip/hip_runtime.h"
#include "log_likelihood.h"

#include "utils/GpuManager.h"
#include "utils/Memory.h"
#include "utils/ScopedTimer.h"

#include <iostream>
#include <set>

/*************** Kernels **********************/

__global__ void calc_LLError_kernel(const unsigned char *mask,
                                    const float *LL,
                                    const float *Idata,
                                    const int *addr_info,
                                    float *LLError,
                                    int m,
                                    int n,
                                    const int *da_unique)
{
  // buffer to sum the matrices in shared memory
  extern __shared__ float sumbuffer[];

  int batch = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int txy = tx * blockDim.y + ty;
  sumbuffer[txy] = 0.0f;

  auto da = addr_info + da_unique[batch] * 3 * 5 + 9;
  auto ma = da + 3;

  auto da0 = da[0];
  auto ma0 = ma[0];
  LL += da0 * m * n;
  Idata += da0 * m * n;
  mask += ma0 * m * n;
  LLError += da0;

  for (int i = tx; i < m; i += blockDim.x)
  {
    for (int j = ty; j < n; j += blockDim.y)
    {
      auto vLL = LL[i * n + j];
      auto vIdata = Idata[i * n + j];
      auto vMask = mask[i * n + j];

      auto m_by_LL_minus_Idata = vMask ? vLL - vIdata : 0.0f;
      auto m_by_LL_minus_Idata_sqr = m_by_LL_minus_Idata * m_by_LL_minus_Idata;

      auto vIdata_p_1 = vIdata + 1.0f;
      auto sumval = m_by_LL_minus_Idata_sqr / vIdata_p_1;
      sumbuffer[txy] += sumval;
    }
  }

  // now add up sumbuffer in shared memory
  __syncthreads();
  int nt = blockDim.x * blockDim.y;
  int c = nt;
  while (c > 1)
  {
    int half = c / 2;
    if (txy < half)
    {
      sumbuffer[txy] += sumbuffer[c - txy - 1];
    }
    __syncthreads();
    c = c - half;
  }

  if (txy == 0)
  {
    auto v = sumbuffer[0] / float(n * m);
    LLError[0] = v;
  }
}

/*************** Class implementation **********/

LogLikelihood::LogLikelihood() : CudaFunction("log_likelihood") {}

void LogLikelihood::setParameters(int i, int m, int n, int addr_i, int Idata_i)
{
  i_ = i;
  m_ = m;
  n_ = n;
  Idata_i_ = Idata_i;  // size of mask as well
  addr_i_ = addr_i;    // this is same as I

  ffprop_ = gpuManager.get_cuda_function<FarfieldPropagator>(
      "loglikelihood.farfield_propagator", i, m, n);
  abs2_ = gpuManager.get_cuda_function<Abs2<complex<float>, float>>(
      "loglikelihood.abs2", i * m * n);
  sum2buffer_ = gpuManager.get_cuda_function<SumToBuffer<float>>(
      "loglikelihood.sum2buffer", i_, m, n, Idata_i_, m, n, addr_i, addr_i);
  sum2buffer_->setAddrStride(5 * 3);
}

void LogLikelihood::setDeviceBuffers(complex<float> *d_probe_obj,
                                     unsigned char *d_mask,
                                     float *d_Idata,
                                     complex<float> *d_prefilter,
                                     complex<float> *d_postfilter,
                                     int *d_addr_info,
                                     float *d_out,
                                     int *d_outidx,
                                     int *d_startidx,
                                     int *d_indices,
                                     int outidx_size)
{
  d_probe_obj_ = d_probe_obj;
  d_mask_ = d_mask;
  d_Idata_ = d_Idata;
  d_prefilter_ = d_prefilter;
  d_postfilter_ = d_postfilter;
  d_addr_info_ = d_addr_info;
  d_out_ = d_out;
  d_outidx_ = d_outidx;
  d_startidx_ = d_startidx;
  d_indices_ = d_indices;
  outidx_size_ = outidx_size;
}

int LogLikelihood::calculateAddrIndices(const int *out1_addr)
{
  outidx_size_ = sum2buffer_->calculateAddrIndices(out1_addr);
  return outidx_size_;
}

void LogLikelihood::calculateUniqueDaIndices(const int *da_addr)
{
  std::vector<int> unique;
  unique.reserve(addr_i_);
  std::set<int> values;

  for (auto i = 0; i < addr_i_; ++i)
  {
    if (values.insert(da_addr[i * 15]).second)
      unique.push_back(i);
  }

  // for (auto i : unique) {
  //  std::cout << i << std::endl;
  //}

  d_da_unique_.allocate(unique.size());
  gpu_memcpy_h2d(d_da_unique_.get(), unique.data(), unique.size());
}

void LogLikelihood::allocate()
{
  ScopedTimer t(this, "allocate");
  d_probe_obj_.allocate(i_ * m_ * n_);
  d_mask_.allocate(Idata_i_ * m_ * n_);
  d_Idata_.allocate(Idata_i_ * m_ * n_);
  d_prefilter_.allocate(m_ * n_);
  d_postfilter_.allocate(m_ * n_);
  d_addr_info_.allocate(addr_i_ * 5 * 3);
  d_out_.allocate(Idata_i_);
  d_LL_.allocate(Idata_i_ * m_ * n_);
  d_ft_.allocate(i_ * m_ * n_);
  d_abs2_ft_.allocate(i_ * m_ * n_);

  ffprop_->setDeviceBuffers(
      d_probe_obj_.get(), d_ft_.get(), d_prefilter_.get(), d_postfilter_.get());
  ffprop_->allocate();

  abs2_->setDeviceBuffers(d_ft_.get(), d_abs2_ft_.get());
  abs2_->allocate();

  sum2buffer_->setDeviceBuffers(d_abs2_ft_.get(),
                                d_LL_.get(),
                                d_addr_info_.get() + 6,
                                d_addr_info_.get() + 9,
                                d_outidx_,
                                d_startidx_,
                                d_indices_,
                                outidx_size_);
  sum2buffer_->allocate();
}

void LogLikelihood::updateErrorOutput(float *d_out) { d_out_ = d_out; }

float *LogLikelihood::getOutput() const { return d_out_.get(); }

void LogLikelihood::transfer_in(const complex<float> *probe_obj,
                                const unsigned char *mask,
                                const float *Idata,
                                const complex<float> *prefilter,
                                const complex<float> *postfilter,
                                const int *addr_info)
{
  ScopedTimer t(this, "transfer in");
  gpu_memcpy_h2d(d_probe_obj_.get(), probe_obj, i_ * m_ * n_);
  gpu_memcpy_h2d(d_mask_.get(), mask, Idata_i_ * m_ * n_);
  gpu_memcpy_h2d(d_Idata_.get(), Idata, Idata_i_ * m_ * n_);
  gpu_memcpy_h2d(d_prefilter_.get(), prefilter, m_ * n_);
  gpu_memcpy_h2d(d_postfilter_.get(), postfilter, m_ * n_);
  // TODO: handle this case more explicitly
  if (!d_addr_info_.isExternal())
  {
    gpu_memcpy_h2d(d_addr_info_.get(), addr_info, addr_i_ * 5 * 3);
  }

  // transfer-in on sum_to_buffer needs to be called, for the internal
  // outidx buffers
  sum2buffer_->transfer_in(nullptr, nullptr, nullptr);

  calculateUniqueDaIndices(addr_info + 9);
}

void LogLikelihood::transfer_out(float *out)
{
  ScopedTimer t(this, "transfer out");
  gpu_memcpy_d2h(out, d_out_.get(), Idata_i_);
}

void LogLikelihood::run()
{
  ScopedTimer t(this, "run");
  ffprop_->run(true, true, true);
  abs2_->run();
  sum2buffer_->run();

  dim3 threadsPerBlock = {32u, 32u, 1u};
  dim3 blocks = {unsigned(d_da_unique_.size()), 1u, 1u};
  calc_LLError_kernel<<<blocks,
                        threadsPerBlock,
                        threadsPerBlock.x * threadsPerBlock.y *
                            sizeof(float)>>>(d_mask_.get(),
                                             d_LL_.get(),
                                             d_Idata_.get(),
                                             d_addr_info_.get(),
                                             d_out_.get(),
                                             m_,
                                             n_,
                                             d_da_unique_.get());
  checkLaunchErrors();

  // sync device if timing is enabled
  timing_sync();
}

extern "C" void log_likelihood_c(const float *fprobe_obj,
                                 const unsigned char *mask,
                                 const float *Idata,
                                 const float *fprefilter,
                                 const float *fpostfilter,
                                 const int *addr_info,
                                 float *out,
                                 int i,
                                 int m,
                                 int n,
                                 int addr_i,
                                 int Idata_i)
{
  auto probe_obj = reinterpret_cast<const complex<float> *>(fprobe_obj);
  auto prefilter = reinterpret_cast<const complex<float> *>(fprefilter);
  auto postfilter = reinterpret_cast<const complex<float> *>(fpostfilter);

  auto ll = gpuManager.get_cuda_function<LogLikelihood>(
      "loglikelihood", i, m, n, addr_i, Idata_i);
  ll->calculateAddrIndices(addr_info + 9);
  ll->allocate();
  ll->transfer_in(probe_obj, mask, Idata, prefilter, postfilter, addr_info);
  ll->run();
  ll->transfer_out(out);
}