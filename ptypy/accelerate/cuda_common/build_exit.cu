#include "hip/hip_runtime.h"
/** build_exit kernel.
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs (float or double)
 * - OUT_TYPE: the data type for the outputs (float or double - for aux wave)
 * - MATH_TYPE: the data type used for computation 
 */


#include <thrust/complex.h>
using thrust::complex;

template <class T>
__device__ inline void atomicAdd(complex<T>* x, complex<T> y)
{
  auto xf = reinterpret_cast<T*>(x);
  atomicAdd(xf, y.real());
  atomicAdd(xf + 1, y.imag());
}

extern "C" __global__ void build_exit(complex<OUT_TYPE>* auxiliary_wave,
                                      complex<OUT_TYPE>* exit_wave,
                                      int B,
                                      int C,
                                      const complex<IN_TYPE>* __restrict__ probe,
                                      int E,
                                      int F,
                                      const complex<IN_TYPE>* __restrict__ obj,
                                      int H,
                                      int I,
                                      const int* __restrict__ addr,
                                      IN_TYPE alpha_)
{
  int bid = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  const int addr_stride = 15;
  const MATH_TYPE alpha = alpha_;  // type conversion

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];
  exit_wave += ea[0] * B * C;
  auxiliary_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
#pragma unroll(4)  // we use blockDim.x = 32, and C is typically more than 128
                   // (it will work for less as well)
    for (int c = tx; c < C; c += blockDim.x)
    {
      complex<MATH_TYPE> auxv = auxiliary_wave[b * C + c];
      complex<MATH_TYPE> t_probe = probe[b * F + c];
      complex<MATH_TYPE> t_obj = obj[b * I + c];
      complex<MATH_TYPE> t_exit = exit_wave[b * C + c];
      auxv -= alpha * t_probe * t_obj;
      auxv += (alpha - 1) * t_exit;
      exit_wave[b * C + c] += auxv;
      auxiliary_wave[b * C + c] = auxv;
    }
  }
}
