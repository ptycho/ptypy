#include "hip/hip_runtime.h"
/** pr_update_ML.
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs (float or double)
 * - OUT_TYPE: the data type for the outputs (float or double)
 * - MATH_TYPE: the data type used for computation
 */


#include "common.cuh"

template <class T>
__device__ inline void atomicAdd(complex<T>* x, const complex<T>& y)
{
  auto xf = reinterpret_cast<T*>(x);
  atomicAdd(xf, y.real());
  atomicAdd(xf + 1, y.imag());
}

extern "C" __global__ void pr_update_ML(const complex<IN_TYPE>* __restrict__ exit_wave,
                                        int A,
                                        int B,
                                        int C,
                                        complex<OUT_TYPE>* probe,
                                        int D,
                                        int E,
                                        int F,
                                        const complex<IN_TYPE>* __restrict__ obj,
                                        int G,
                                        int H,
                                        int I,
                                        const int* __restrict__ addr,
                                        IN_TYPE fac_)
{
  assert(B == E);  // prsh[1]
  assert(C == F);  // prsh[2]
  const int bid = blockIdx.x;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int addr_stride = 15;
  MATH_TYPE fac = fac_;

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];

  assert(oa[0] * H * I + oa[1] * I + oa[2] + (B - 1) * I + C - 1 < G * H * I);

  exit_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
    for (int c = tx; c < C; c += blockDim.x)
    {
      complex<MATH_TYPE> obj_val = obj[b * I + c];
      complex<MATH_TYPE> exit_val = exit_wave[b * C + c];
      complex<MATH_TYPE> add_val_m = conj(obj_val) * exit_val * fac;
      complex<OUT_TYPE> add_val = add_val_m;
      atomicAdd(&probe[b * F + c], add_val);
    }
  }
}
