#include "hip/hip_runtime.h"
/** ob_update_ML.
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs (float or double)
 * - OUT_TYPE: the data type for the outputs (float or double)
 * - MATH_TYPE: the data type used for computation
 */

#include "common.cuh"

template <class T>
__device__ inline void atomicAdd(complex<T>* x, const complex<T>& y)
{
  auto xf = reinterpret_cast<T*>(x);
  atomicAdd(xf, y.real());
  atomicAdd(xf + 1, y.imag());
}

extern "C"
{
  __global__ void ob_update_ML(const complex<IN_TYPE>* __restrict__ exit_wave,
                               int A,
                               int B,
                               int C,
                               const complex<IN_TYPE>* __restrict__ probe,
                               int D,
                               int E,
                               int F,
                               complex<OUT_TYPE>* obj,
                               int G,
                               int H,
                               int I,
                               const int* __restrict__ addr,
                               IN_TYPE fac_)
  {
    const int bid = blockIdx.x;
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int addr_stride = 15;
    MATH_TYPE fac = fac_;

    const int* oa = addr + 3 + bid * addr_stride;
    const int* pa = addr + bid * addr_stride;
    const int* ea = addr + 6 + bid * addr_stride;

    probe += pa[0] * E * F + pa[1] * F + pa[2];
    obj += oa[0] * H * I + oa[1] * I + oa[2];

    assert(oa[0] * H * I + oa[1] * I + oa[2] + (B - 1) * I + C - 1 < G * H * I);

    exit_wave += ea[0] * B * C;

    for (int b = ty; b < B; b += blockDim.y)
    {
      for (int c = tx; c < C; c += blockDim.x)
      {
        complex<MATH_TYPE> probe_val = probe[b * F + c];
        complex<MATH_TYPE> exit_val = exit_wave[b * C + c];
        complex<MATH_TYPE> add_val_m = conj(probe_val) * exit_val * fac;
        complex<OUT_TYPE> add_val(add_val_m);

        atomicAdd(&obj[b * I + c], add_val);
      }
    }
  }
}