#include "hip/hip_runtime.h"
/** log_likelihood kernel.
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs (float or double)
 * - OUT_TYPE: the data type for the outputs (float or double)
 * - MATH_TYPE: the data type used for computation
 */

#include "common.cuh"

// specify max number of threads/block and min number of blocks per SM,
// to assist the compiler in register optimisations.
// We achieve a higher occupancy in this case, as less registers are used
// (guided by profiler)
extern "C" __global__ void __launch_bounds__(1024, 2)
    log_likelihood(int nmodes,
                   complex<OUT_TYPE> *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int addr_stride = 15;

  const int *ea = addr + 6 + (blockIdx.x * nmodes) * addr_stride;
  const int *da = addr + 9 + (blockIdx.x * nmodes) * addr_stride;
  const int *ma = addr + 12 + (blockIdx.x * nmodes) * addr_stride;

  aux += ea[0] * A * B;
  fmag += da[0] * A * B;
  fmask += ma[0] * A * B;
  llerr += da[0] * A * B;
  MATH_TYPE norm = A * B;

  for (int a = ty; a < A; a += blockDim.y)
  {
    for (int b = tx; b < B; b += blockDim.x)
    {
      MATH_TYPE acc = 0.0;
      for (int idx = 0; idx < nmodes; ++idx)
      {
        complex<MATH_TYPE> t_aux = aux[a * B + b + idx * A * B];
        MATH_TYPE abs_exit_wave = abs(t_aux);
        acc += abs_exit_wave *
               abs_exit_wave;  // if we do this manually (real*real +imag*imag)
                               // we get differences to numpy due to rounding
      }
      auto I = MATH_TYPE(fmag[a * B + b]) * MATH_TYPE(fmag[a * B + b]);
      llerr[a * B + b] =
          MATH_TYPE(fmask[a * B + b]) * (acc - I) * (acc - I) / (I + 1) / norm;
    }
  }
}


extern "C" __global__ void 
    log_likelihood2(int nmodes,
                   complex<OUT_TYPE> *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  int bid = blockIdx.z;
  int tx = threadIdx.x;
  int a = threadIdx.y + blockIdx.y * blockDim.y;
  if (a >= A)
    return;
  int addr_stride = 15;

  const int *ea = addr + 6 + (bid * nmodes) * addr_stride;
  const int *da = addr + 9 + (bid * nmodes) * addr_stride;
  const int *ma = addr + 12 + (bid * nmodes) * addr_stride;

  aux += ea[0] * A * B;
  fmag += da[0] * A * B;
  fmask += ma[0] * A * B;
  llerr += da[0] * A * B;
  MATH_TYPE norm = A * B;

  for (int b = tx; b < B; b += blockDim.x)
  {
    MATH_TYPE acc = 0.0;
    for (int idx = 0; idx < nmodes; ++idx)
    {
      complex<MATH_TYPE> t_aux = aux[a * B + b + idx * A * B];
      MATH_TYPE abs_exit_wave = abs(t_aux);
      acc += abs_exit_wave *
              abs_exit_wave;  // if we do this manually (real*real +imag*imag)
                              // we get differences to numpy due to rounding
    }
    auto I = MATH_TYPE(fmag[a * B + b]) * MATH_TYPE(fmag[a * B + b]);
    llerr[a * B + b] =
        MATH_TYPE(fmask[a * B + b]) * (acc - I) * (acc - I) / (I + 1) / norm;
  }
}

// ML variant which uses weights and intensity directly.
// Based of log_likelihood
extern "C" __global__ void __launch_bounds__(1024, 2)
    log_likelihood_ml(int nmodes,
                   complex<OUT_TYPE> *aux,
                   const IN_TYPE *weights,
                   const IN_TYPE *I,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int addr_stride = 15;

  const int *ea = addr + 6 + (blockIdx.x * nmodes) * addr_stride;
  const int *da = addr + 9 + (blockIdx.x * nmodes) * addr_stride;
  const int *ma = addr + 12 + (blockIdx.x * nmodes) * addr_stride;

  aux += ea[0] * A * B;
  I += da[0] * A * B;
  weights += ma[0] * A * B;
  llerr += da[0] * A * B;
  MATH_TYPE norm = A * B;

  for (int a = ty; a < A; a += blockDim.y)
  {
    for (int b = tx; b < B; b += blockDim.x)
    {
      MATH_TYPE acc = 0.0;
      MATH_TYPE i = I[a * B + b];
      for (int idx = 0; idx < nmodes; ++idx)
      {
        complex<MATH_TYPE> t_aux = aux[a * B + b + idx * A * B];
        MATH_TYPE abs_exit_wave = abs(t_aux);
        acc += abs_exit_wave *
               abs_exit_wave;  // if we do this manually (real*real +imag*imag)
                               // we get differences to numpy due to rounding
      }
      llerr[a * B + b] =
          MATH_TYPE(weights[a * B + b]) * (acc - i) * (acc - i) / norm;
    }
  }
}