#include "hip/hip_runtime.h"
/** difference along last axis
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs 
 * - OUT_TYPE: the data type for the outputs 
 */

#include <thrust/complex.h>
using thrust::complex;

/** This is the special case for when we diff along the last axis.
 * 
 * Here, flat_dim is all other dims multiplied together, and axis_dim
 * is the dimension along which we diff. 
 * To ensure that we stay coalesced (compared to delx_mid), 
 * we use the x index to iterate within each thread block (the loop).
 * Otherwise it follows the same ideas as delx_mid - please read the
 * description there.
  */
extern "C" __global__ void delx_last(const IN_TYPE *__restrict__ input,
                                     OUT_TYPE *output,
                                     int flat_dim,
                                     int axis_dim)
{
  // reinterpret to avoid constructor of complex<float>() + compiler warning
  __shared__ char shr[BDIM_X * BDIM_Y * sizeof(IN_TYPE)];
  auto shared_data = reinterpret_cast<IN_TYPE *>(shr);

  unsigned int tx = threadIdx.x;
  unsigned int ty = threadIdx.y;

  unsigned int ix = tx;
  unsigned int iy = ty + blockIdx.x * BDIM_Y;  // we always use x in grid

  int stride_y = axis_dim;

  auto maxblocks = (axis_dim + BDIM_X - 1) / BDIM_X;
  for (int bidx = 0; bidx < maxblocks; ++bidx)
  {
    ix = tx + bidx * BDIM_X;

    if (iy < flat_dim && ix < axis_dim)
    {
      shared_data[ty * BDIM_X + tx] = input[iy * stride_y + ix];
    }

    __syncthreads();

    if (iy < flat_dim && ix < axis_dim)
    {
      if (IS_FORWARD)
      {
        IN_TYPE plus1;
        if (tx < BDIM_X - 1 &&
            ix < axis_dim - 1)  // we have a next element in shared data
        {
          plus1 = shared_data[ty * BDIM_X + tx + 1];
        }
        else if (ix == axis_dim - 1)  // end of axis - same as current to get 0
        {
          plus1 = shared_data[ty * BDIM_X + tx];
        }
        else  // end of block, but nore input is there
        {
          plus1 = input[iy * stride_y + ix + 1];
        }

        output[iy * stride_y + ix] = plus1 - shared_data[ty * BDIM_X + tx];
      }
      else
      {
        IN_TYPE minus1;
        if (tx > 0)  // we have a previous element in shared
        {
          minus1 = shared_data[ty * BDIM_X + tx - 1];
        }
        else if (ix == 0)  // use same as next to get zero
        {
          minus1 = shared_data[ty * BDIM_X + tx];
        }
        else  // read previous input (ty == 0 but iy > 0)
        {
          minus1 = input[iy * stride_y + ix - 1];
        }
        output[iy * stride_y + ix] = shared_data[ty * BDIM_X + tx] - minus1;
      }
    }
  }
}