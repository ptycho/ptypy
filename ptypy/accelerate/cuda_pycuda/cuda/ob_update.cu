#include "hip/hip_runtime.h"
#include <thrust/complex.h>
using thrust::complex;

template <class T>
__device__ inline void atomicAdd(complex<T>* x, complex<T> y)
{
  auto xf = reinterpret_cast<T*>(x);
  atomicAdd(xf, y.real());
  atomicAdd(xf + 1, y.imag());
}

extern "C" __global__ void ob_update(
    const complex<float>* __restrict__ exit_wave,
    int A,
    int B,
    int C,
    const complex<float>* __restrict__ probe,
    int D,
    int E,
    int F,
    complex<float>* obj,
    int G,
    int H,
    int I,
    const int* __restrict__ addr,
    DENOM_TYPE* denominator)
{
  const int bid = blockIdx.x;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int addr_stride = 15;

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];
  denominator += oa[0] * H * I + oa[1] * I + oa[2];

  assert(oa[0] * H * I + oa[1] * I + oa[2] + (B - 1) * I + C - 1 < G * H * I);

  exit_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
    for (int c = tx; c < C; c += blockDim.x)
    {
      auto probe_val = probe[b * F + c];
      atomicAdd(&obj[b * I + c], conj(probe_val) * exit_wave[b * C + c]);
      auto denomreal = reinterpret_cast<float*>(&denominator[b * I + c]);
      auto upd_probe = probe_val.real() * probe_val.real() +
                       probe_val.imag() * probe_val.imag();
      atomicAdd(denomreal, upd_probe);
    }
  }
}
