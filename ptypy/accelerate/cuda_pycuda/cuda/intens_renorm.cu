#include "hip/hip_runtime.h"
#include <thrust/complex.h>
using thrust::complex;

extern "C" __global__ void step1(const FTYPE* Imodel,
                                   const FTYPE* I,
                                   const FTYPE* w,
                                   FTYPE* num,
                                   FTYPE* den,
                                   int z,
                                   int x)
{
  int iz = blockIdx.z;
  int ix = threadIdx.x + blockIdx.x * blockDim.x;

  if (iz >= z || ix >= x)
    return;

  auto tmp = w[iz * x + ix] * Imodel[iz * x + ix];
  num[iz * x + ix] = tmp * I[iz * x + ix];
  den[iz * x + ix] = tmp * Imodel[iz * x + ix];
}

extern "C" __global__ void step2(const FTYPE* fic_tmp,
                                 FTYPE* fic,
                                 FTYPE* Imodel,
                                 int z,
                                 int x)
{
  int iz = blockIdx.z;
  int ix = threadIdx.x + blockIdx.x * blockDim.x;

  if (iz >= z || ix >= x)
    return;
  //probably not so clever having all threads read from the same locations
  auto tmp = fic[iz] / fic_tmp[iz];
  Imodel[iz * x + ix] *= tmp;
  // race condition if write is not restricted to one thread
  // learned this the hard way
  if (ix==0)
    fic[iz] = tmp;
}