#include "hip/hip_runtime.h"
#include <thrust/complex.h>
using thrust::complex;

template <class T>
__device__ inline void atomicAdd(complex<T>* x, complex<T> y)
{
  auto xf = reinterpret_cast<T*>(x);
  atomicAdd(xf, y.real());
  atomicAdd(xf + 1, y.imag());
}

extern "C" __global__ void pr_update_ML(const CTYPE* __restrict__ exit_wave,
                                        int A,
                                        int B,
                                        int C,
                                        CTYPE* probe,
                                        int D,
                                        int E,
                                        int F,
                                        const CTYPE* __restrict__ obj,
                                        int G,
                                        int H,
                                        int I,
                                        const int* __restrict__ addr,
                                        FTYPE fac)
{
  assert(B == E);  // prsh[1]
  assert(C == F);  // prsh[2]
  const int bid = blockIdx.x;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int addr_stride = 15;

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];

  assert(oa[0] * H * I + oa[1] * I + oa[2] + (B - 1) * I + C - 1 < G * H * I);

  exit_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
    for (int c = tx; c < C; c += blockDim.x)
    {
      auto obj_val = obj[b * I + c];
      atomicAdd(&probe[b * F + c], conj(obj_val) * exit_wave[b * C + c] * fac);
    }
  }
}
