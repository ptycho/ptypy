#include "hip/hip_runtime.h"
#include <thrust/complex.h>
using thrust::complex;

extern "C" __global__ void build_aux(
    complex<float>* auxiliary_wave,
    const complex<float>* __restrict__ exit_wave,
    int B,
    int C,
    const complex<float>* __restrict__ probe,
    int E,
    int F,
    const complex<float>* __restrict__ obj,
    int H,
    int I,
    const int* __restrict__ addr,
    float alpha)
{
  int bid = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int addr_stride = 15;

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];
  exit_wave += ea[0] * B * C;
  auxiliary_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
#pragma unroll(4)  // we use blockDim.x = 32, and C is typically more than 128
                   // (it will work for less as well)
    for (int c = tx; c < C; c += blockDim.x)
    {
      auxiliary_wave[b * C + c] =
          obj[b * I + c] * probe[b * F + c] * (1.0f + alpha) -
          exit_wave[b * C + c] * alpha;
    }
  }
}
