#include "hip/hip_runtime.h"
/** build_aux kernel.
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs (float or double)
 * - OUT_TYPE: the data type for the outputs (float or double - for aux wave)
 * - MATH_TYPE: the data type used for computation
 */

#include <thrust/complex.h>
using thrust::complex;

// core calculation function - used by both kernels and inlined
inline __device__ complex<MATH_TYPE> calculate(
    const complex<MATH_TYPE>& t_obj,
    const complex<MATH_TYPE>& t_probe,
    const complex<MATH_TYPE>& t_ex,
    MATH_TYPE alpha)
{
  return t_obj * t_probe * (MATH_TYPE(1) + alpha) - t_ex * alpha;
}

extern "C" __global__ void build_aux(
    complex<OUT_TYPE>* auxiliary_wave,
    const complex<IN_TYPE>* __restrict__ exit_wave,
    int B,
    int C,
    const complex<IN_TYPE>* __restrict__ probe,
    int E,
    int F,
    const complex<IN_TYPE>* __restrict__ obj,
    int H,
    int I,
    const int* __restrict__ addr,
    IN_TYPE alpha_)
{
  int bid = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int addr_stride = 15;
  const MATH_TYPE alpha = alpha_;  // type conversion

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];
  exit_wave += ea[0] * B * C;
  auxiliary_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
#pragma unroll(4)  // we use blockDim.x = 32, and C is typically more than 128
                   // (it will work for less as well)
    for (int c = tx; c < C; c += blockDim.x)
    {
      auxiliary_wave[b * C + c] = calculate(
          obj[b * I + c], probe[b * F + c], exit_wave[b * C + c], alpha);
    }
  }
}

extern "C" __global__ void build_aux2(
    complex<OUT_TYPE>* auxiliary_wave,
    const complex<IN_TYPE>* __restrict__ exit_wave,
    int B,
    int C,
    const complex<IN_TYPE>* __restrict__ probe,
    int E,
    int F,
    const complex<IN_TYPE>* __restrict__ obj,
    int H,
    int I,
    const int* __restrict__ addr,
    IN_TYPE alpha_)
{
  int bid = blockIdx.z;
  int tx = threadIdx.x;
  int b = threadIdx.y + blockIdx.y * blockDim.y;
  if (b >= B)
    return;
  int addr_stride = 15;
  const MATH_TYPE alpha = alpha_;  // type conversion

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];
  exit_wave += ea[0] * B * C;
  auxiliary_wave += ea[0] * B * C;

  for (int c = tx; c < C; c += blockDim.x)
  {
    auxiliary_wave[b * C + c] = calculate(
        obj[b * I + c], probe[b * F + c], exit_wave[b * C + c], alpha);
  }
}
