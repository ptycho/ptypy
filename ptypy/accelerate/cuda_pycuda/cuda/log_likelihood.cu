#include "hip/hip_runtime.h"
/** log_likelihood kernel.
 *
 * Data types:
 * - IN_TYPE: the data type for the inputs (float or double)
 * - OUT_TYPE: the data type for the outputs (float or double)
 * - MATH_TYPE: the data type used for computation
 */

#include <cassert>
#include <cmath>
#include <thrust/complex.h>
using std::sqrt;
using thrust::abs;
using thrust::complex;

// version if input is complex, i.e. we need to calc abs(.)**2
inline __device__ MATH_TYPE aux_intensity(complex<MATH_TYPE> aux_t) {
  MATH_TYPE abst = abs(aux_t);
  return abst * abst; // if we do this manually (real*real +imag*imag)
                      // we get differences to numpy due to rounding
}

// version if input is real, so we can just return it
inline __device__ MATH_TYPE aux_intensity(MATH_TYPE aux_t) {
  return aux_t;
}

////////// log_likelihood with 1 thread block per image

template <class AUX_T>
inline __device__ void log_likelihood_impl(
                   int nmodes,
                   AUX_T *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int addr_stride = 15;

  const int *ea = addr + 6 + (blockIdx.x * nmodes) * addr_stride;
  const int *da = addr + 9 + (blockIdx.x * nmodes) * addr_stride;
  const int *ma = addr + 12 + (blockIdx.x * nmodes) * addr_stride;

  aux += ea[0] * A * B;
  fmag += da[0] * A * B;
  fmask += ma[0] * A * B;
  llerr += da[0] * A * B;
  MATH_TYPE norm = A * B;

  for (int a = ty; a < A; a += blockDim.y)
  {
    for (int b = tx; b < B; b += blockDim.x)
    {
      MATH_TYPE acc = 0.0;
      for (int idx = 0; idx < nmodes; ++idx)
      {
        acc += aux_intensity(aux[a * B + b + idx * A * B]);
      }
      auto I = MATH_TYPE(fmag[a * B + b]) * MATH_TYPE(fmag[a * B + b]);
      llerr[a * B + b] =
          MATH_TYPE(fmask[a * B + b]) * (acc - I) * (acc - I) / (I + 1) / norm;
    }
  }
}

// specify max number of threads/block and min number of blocks per SM,
// to assist the compiler in register optimisations.
// We achieve a higher occupancy in this case, as less registers are used
// (guided by profiler)
extern "C" __global__ void __launch_bounds__(1024, 2)
    log_likelihood(int nmodes,
                   complex<OUT_TYPE> *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  log_likelihood_impl(nmodes, aux, fmask, fmag, addr, llerr, A, B);
}

extern "C" __global__ void __launch_bounds__(1024, 2)
    log_likelihood_auxintensity(int nmodes,
                   OUT_TYPE *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  log_likelihood_impl(nmodes, aux, fmask, fmag, addr, llerr, A, B);
}

/////////////////// version with 1 thread block per x dimension only
template <class AUX_T>
__device__ inline void log_likelihood2_impl(
                   int nmodes,
                   AUX_T *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  int bid = blockIdx.z;
  int tx = threadIdx.x;
  int a = threadIdx.y + blockIdx.y * blockDim.y;
  if (a >= A)
    return;
  int addr_stride = 15;

  const int *ea = addr + 6 + (bid * nmodes) * addr_stride;
  const int *da = addr + 9 + (bid * nmodes) * addr_stride;
  const int *ma = addr + 12 + (bid * nmodes) * addr_stride;

  aux += ea[0] * A * B;
  fmag += da[0] * A * B;
  fmask += ma[0] * A * B;
  llerr += da[0] * A * B;
  MATH_TYPE norm = A * B;

  for (int b = tx; b < B; b += blockDim.x)
  {
    MATH_TYPE acc = 0.0;
    for (int idx = 0; idx < nmodes; ++idx)
    {
      acc += aux_intensity(aux[a * B + b + idx * A * B]);
    }
    auto I = MATH_TYPE(fmag[a * B + b]) * MATH_TYPE(fmag[a * B + b]);
    llerr[a * B + b] =
        MATH_TYPE(fmask[a * B + b]) * (acc - I) * (acc - I) / (I + 1) / norm;
  }
}


extern "C" __global__ void 
    log_likelihood2(int nmodes,
                   complex<OUT_TYPE> *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  log_likelihood2_impl(nmodes, aux, fmask, fmag, addr, llerr, A, B);
}

extern "C" __global__ void 
    log_likelihood2_auxintensity(int nmodes,
                   OUT_TYPE *aux,
                   const IN_TYPE *fmask,
                   const IN_TYPE *fmag,
                   const int *addr,
                   IN_TYPE *llerr,
                   int A,
                   int B)
{
  log_likelihood2_impl(nmodes, aux, fmask, fmag, addr, llerr, A, B);
}