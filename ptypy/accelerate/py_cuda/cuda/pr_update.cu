#include "hip/hip_runtime.h"
#include <thrust/complex.h>
using thrust::complex;

template <class T>
__device__ inline void atomicAdd(complex<T>* x, complex<T> y)
{
  auto xf = reinterpret_cast<T*>(x);
  atomicAdd(xf, y.real());
  atomicAdd(xf + 1, y.imag());
}

extern "C" __global__ void pr_update(
    const complex<float>* __restrict__ exit_wave,
    int A,
    int B,
    int C,
    complex<float>* probe,
    int D,
    int E,
    int F,
    const complex<float>* __restrict__ obj,
    int G,
    int H,
    int I,
    const int* __restrict__ addr,
    DENOM_TYPE* denominator)
{
  assert(B == E);  // prsh[1]
  assert(C == F);  // prsh[2]
  const int bid = blockIdx.x;
  const int tx = threadIdx.x;
  const int ty = threadIdx.y;
  const int addr_stride = 15;

  const int* oa = addr + 3 + bid * addr_stride;
  const int* pa = addr + bid * addr_stride;
  const int* ea = addr + 6 + bid * addr_stride;

  probe += pa[0] * E * F + pa[1] * F + pa[2];
  obj += oa[0] * H * I + oa[1] * I + oa[2];
  denominator += pa[0] * E * F + pa[1] * F + pa[2];

  assert(oa[0] * H * I + oa[1] * I + oa[2] + (B - 1) * I + C - 1 < G * H * I);

  exit_wave += ea[0] * B * C;

  for (int b = ty; b < B; b += blockDim.y)
  {
    for (int c = tx; c < C; c += blockDim.x)
    {
      auto obj_val = obj[b * I + c];
      atomicAdd(&probe[b * F + c], conj(obj_val) * exit_wave[b * C + c]);
      auto denomreal = reinterpret_cast<float*>(&denominator[b * F + c]);
      auto upd_obj =
          obj_val.real() * obj_val.real() + obj_val.imag() * obj_val.imag();
      atomicAdd(denomreal, upd_obj);
    }
  }
}
