#include "hip/hip_runtime.h"
#include <cassert>
#include <thrust/complex.h>
using thrust::complex;

extern "C" __global__ void update_addr_error_state(int* addr,
                                                   const int* mangled_addr,
                                                   float* error_state,
                                                   const float* error_sum,
                                                   int nmodes)
{
  int tx = threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  // we're using one warp only in x direction, to get implicit
  // intra-warp sync between reading err_st and writing it
  assert(blockDim.x <= 32);

  addr += row * nmodes * 15;
  mangled_addr += row * nmodes * 15;

  auto err_sum = error_sum[row];
  auto err_st = error_state[row];

  if (err_sum < err_st)
  {
    for (int i = tx; i < nmodes * 15; i += blockDim.x)
    {
      addr[i] = mangled_addr[i];
    }
  }

  if (tx == 0 && err_sum < err_st)
  {
    error_state[row] = error_sum[row];
  }
}